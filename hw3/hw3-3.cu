#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cassert>
#include <zlib.h>
#include <math.h>
#include <iomanip>  // for std::setw
#include <cstdio>   // for fread
#include <omp.h>


#define NOT_REACHABLE         (1073741823)
#define BASIC_WARP            (32)
#define COALESCED_FACTOR      (BASIC_WARP*4)
#define BLOCKING_FACTOR   (BASIC_WARP*2)
// change this to change thread setting
#define THREAD_XY_INTERACTION (2)
// THREAD_X_DIM * THREAD_Y_DIM = 1024
#define THREAD_X_DIM          (BASIC_WARP/THREAD_XY_INTERACTION)
#define THREAD_Y_DIM          (BASIC_WARP*THREAD_XY_INTERACTION)
#define THREAD_LOAD_SAVE_NUM  (BLOCKING_FACTOR/THREAD_X_DIM)

using namespace std;

int maxWidth;
int num_vertex, num_edge, num_blocked_square_row, num_blocked_square_col;
int padded_num_vertex; // extend to fit COALESCED_FACTOR
int *dist; // one dimension array, lenght = padded_num_vertex * padded_num_vertex
int *rem_orig_dist;

void input(char* input_file) {
    FILE* file = fopen(input_file, "rb");
    fread(&num_vertex, sizeof(int), 1, file);
    fread(&num_edge, sizeof(int), 1, file);
    // pad the matrix to fit COALESCED_FACTOR
    int remainder = num_vertex % COALESCED_FACTOR;
    padded_num_vertex = (remainder == 0) ? num_vertex : num_vertex + (COALESCED_FACTOR - remainder);
    dist = (int*)malloc(padded_num_vertex * padded_num_vertex * sizeof(int));
    //rem_orig_dist = (int*)malloc(padded_num_vertex * padded_num_vertex * sizeof(int));
    // initialize dist
    for (int i = 0; i < padded_num_vertex; ++i){ 
        for (int j = 0; j < padded_num_vertex; ++j){
            dist[i * padded_num_vertex + j] = (i == j) ? 0 : NOT_REACHABLE;
            //rem_orig_dist[i * padded_num_vertex + j] = (i == j) ? 0 : NOT_REACHABLE;
        }
    }
    // read file
    int pair[3];
    for (int i = 0; i < num_edge; ++i) {
        fread(pair, sizeof(int), 3, file);
        int from = pair[0], to = pair[1], weight = pair[2];
        dist[from * padded_num_vertex + to] = weight;
        //rem_orig_dist[from * padded_num_vertex + to] = weight;
    }
    fclose(file);
}
void output(char* output_file) {
    FILE* outfile = fopen(output_file, "w");
    // unpad the matrix
    for (int i = 0; i < num_vertex; ++i) 
        for (int j = 0; j < num_vertex; ++j)
            // since we padded the matrix, we have to move the result back
            // since we travel the matrix in row-major order, we will not mess up the result by using the stride of padded_num_vertex
            dist[i * num_vertex + j] = (dist[i * padded_num_vertex + j] >= NOT_REACHABLE) ? NOT_REACHABLE : dist[i * padded_num_vertex + j];
    // use fwrite to speed up output
    fwrite(dist, sizeof(int), num_vertex * num_vertex, outfile);
    fclose(outfile);
}

__global__ void phase_1(int* d_dist, int padded_num_vertex, int round){
    int glb_block_i = round;
    int glb_block_j = round;
    int in_block_i = threadIdx.y;
    int in_block_j = threadIdx.x * THREAD_LOAD_SAVE_NUM;
    // load gloval data into my basic block's share memory
    extern __shared__ int share_target[BLOCKING_FACTOR][BLOCKING_FACTOR];
    #pragma unroll 4
    for(int id = 0; id < THREAD_LOAD_SAVE_NUM; id++){
        share_target[in_block_i][in_block_j + id] =  d_dist[(glb_block_i*BLOCKING_FACTOR+in_block_i) * padded_num_vertex + (glb_block_j * BLOCKING_FACTOR + in_block_j) + id];
    }
    __syncthreads();
    // start computing and enumerate k
    for(int k=0;k<BLOCKING_FACTOR;k++){
        #pragma unroll 4
        for(int id = 0; id<THREAD_LOAD_SAVE_NUM; id++){
            //new from in_block_i to in_block_j+id          old from in_block_i to in_block_j+id                    from  in_block_i to k, from k to in_block_j+id
            share_target[in_block_i][in_block_j + id] = min(
                share_target[in_block_i][in_block_j + id],
                share_target[in_block_i][k] + share_target[k][in_block_j + id]);
        }
        // there are dependence inbetween k at phase 1
        __syncthreads();
    }
    // write back
    #pragma unroll 4
    for(int id = 0; id < THREAD_LOAD_SAVE_NUM; id++){
        d_dist[(glb_block_i*BLOCKING_FACTOR+in_block_i) * padded_num_vertex + (glb_block_j * BLOCKING_FACTOR + in_block_j) + id] = share_target[in_block_i][in_block_j + id];
    }
}

__global__ void phase_2_fuse(int* d_dist, int padded_num_vertex, int round){
    if(round == blockIdx.x) return;
    int glb_b_i_same_row = round;
    int glb_b_j_same_row = blockIdx.x;
    int glb_b_i_same_col = blockIdx.x;
    int glb_b_j_same_col = round;
    int in_block_i = threadIdx.y;
    int in_block_j = threadIdx.x * THREAD_LOAD_SAVE_NUM;

    // load data
    extern __shared__ int from_same_row[BLOCKING_FACTOR][BLOCKING_FACTOR];
    extern __shared__ int from_same_col[BLOCKING_FACTOR][BLOCKING_FACTOR];
    extern __shared__ int to[BLOCKING_FACTOR][BLOCKING_FACTOR];

    for(int id=0;id<THREAD_LOAD_SAVE_NUM;id++){
        from_same_row[in_block_i][in_block_j+id] = d_dist[(glb_b_i_same_row*BLOCKING_FACTOR+in_block_i)*padded_num_vertex + (glb_b_j_same_row*BLOCKING_FACTOR + in_block_j) + id];
        from_same_col[in_block_i][in_block_j+id] = d_dist[(glb_b_i_same_col*BLOCKING_FACTOR+in_block_i)*padded_num_vertex+(glb_b_j_same_col*BLOCKING_FACTOR+in_block_j) + id];
        to[in_block_i][in_block_j+id]   = d_dist[(round*BLOCKING_FACTOR+in_block_i)* padded_num_vertex + (round*BLOCKING_FACTOR+in_block_j) + id];
    }
    __syncthreads();
    for(int k=0;k<BLOCKING_FACTOR;k++){
        #pragma unroll 4
        for(int id=0;id<THREAD_LOAD_SAVE_NUM;id++){
            from_same_row[in_block_i][in_block_j+id] = min(
                from_same_row[in_block_i][in_block_j+id],
                to[in_block_i][k] + from_same_row[k][in_block_j+id]
            );
            from_same_col[in_block_i][in_block_j+id] = min(
                from_same_col[in_block_i][in_block_j+id],
                from_same_col[in_block_i][k] + to[k][in_block_j+id]
            );
        }
        __syncthreads();
    }
    for(int id=0;id < THREAD_LOAD_SAVE_NUM;id++){
        d_dist[(glb_b_i_same_row*BLOCKING_FACTOR + in_block_i) * padded_num_vertex + (glb_b_j_same_row*BLOCKING_FACTOR+in_block_j) + id] = from_same_row[in_block_i][in_block_j+id];
        d_dist[(glb_b_i_same_col*BLOCKING_FACTOR + in_block_i) * padded_num_vertex + (glb_b_j_same_col*BLOCKING_FACTOR+in_block_j) + id] = from_same_col[in_block_i][in_block_j+id];
    }
}

__global__ void phase_2_same_row(int* d_dist, int padded_num_vertex, int round){
    // for now, only ha
    if(round == blockIdx.x) return;
    int glb_block_i = round;
    int glb_block_j = blockIdx.x;
    int in_block_i = threadIdx.y;
    int in_block_j = threadIdx.x * THREAD_LOAD_SAVE_NUM;

    // load data
    extern __shared__ int from[BLOCKING_FACTOR][BLOCKING_FACTOR];
    extern __shared__ int to[BLOCKING_FACTOR][BLOCKING_FACTOR];
    #pragma unroll 4
    for(int id=0;id<THREAD_LOAD_SAVE_NUM;id++){
        from[in_block_i][in_block_j+id] = d_dist[(glb_block_i*BLOCKING_FACTOR+in_block_i)*padded_num_vertex + (glb_block_j*BLOCKING_FACTOR + in_block_j) + id];
        to[in_block_i][in_block_j+id]   = d_dist[(round*BLOCKING_FACTOR+in_block_i)* padded_num_vertex + (round*BLOCKING_FACTOR+in_block_j) + id];
    }
    __syncthreads();

    // cal
    for(int k=0;k<BLOCKING_FACTOR;k++){
        #pragma unroll 4
        for(int id=0;id<THREAD_LOAD_SAVE_NUM;id++){
            from[in_block_i][in_block_j+id] = min(
                from[in_block_i][in_block_j+id],
                to[in_block_i][k] + from[k][in_block_j+id]
            );
        }
        __syncthreads();
    }
    #pragma unroll 4
    for(int id=0;id < THREAD_LOAD_SAVE_NUM;id++){
        d_dist[(glb_block_i*BLOCKING_FACTOR + in_block_i) * padded_num_vertex + (glb_block_j*BLOCKING_FACTOR+in_block_j) + id] = from[in_block_i][in_block_j+id];
    }
}
__global__ void phase_2_same_col(int* d_dist, int padded_num_vertex, int round){
    if(round == blockIdx.x) return;
    int glb_block_i = blockIdx.x;
    int glb_block_j = round;
    int in_block_i = threadIdx.y;
    int in_block_j = threadIdx.x * THREAD_LOAD_SAVE_NUM;
    // load data
    extern __shared__ int from[BLOCKING_FACTOR][BLOCKING_FACTOR];
    extern __shared__ int to[BLOCKING_FACTOR][BLOCKING_FACTOR];
    #pragma unroll 4
    for(int id=0;id<THREAD_LOAD_SAVE_NUM;id++){
        from[in_block_i][in_block_j+id]   = d_dist[(glb_block_i*BLOCKING_FACTOR+in_block_i)*padded_num_vertex+(glb_block_j*BLOCKING_FACTOR+in_block_j) + id];
        to[in_block_i][in_block_j+id] = d_dist[(round*BLOCKING_FACTOR+in_block_i)*padded_num_vertex + (round*BLOCKING_FACTOR+in_block_j)+ id];
    }
    __syncthreads();
    
    for(int k=0;k<BLOCKING_FACTOR;k++){
        #pragma unroll 4
        for(int id=0;id<THREAD_LOAD_SAVE_NUM;id++){
            from[in_block_i][in_block_j+id] = min(
                from[in_block_i][in_block_j+id],
                from[in_block_i][k] + to[k][in_block_j+id]
            );
        }
        __syncthreads();
    }
    #pragma unroll 4
    for(int id=0;id<THREAD_LOAD_SAVE_NUM;id++){
        d_dist[(glb_block_i*BLOCKING_FACTOR + in_block_i) * padded_num_vertex + (glb_block_j*BLOCKING_FACTOR+in_block_j) + id] = from[in_block_i][in_block_j+id];
    }
}

__global__ void phase_3(int* d_dist, int padded_num_vertex, int round, int offset){
    if(round == blockIdx.x || round == (blockIdx.y+offset/BLOCKING_FACTOR)) return;
    int glb_block_i = blockIdx.y;
    int glb_block_j = blockIdx.x;
    int in_block_i = threadIdx.y;
    int in_block_j = threadIdx.x * THREAD_LOAD_SAVE_NUM;
    int arr[THREAD_LOAD_SAVE_NUM];
    extern __shared__ int from[BLOCKING_FACTOR][BLOCKING_FACTOR];
    extern __shared__ int to[BLOCKING_FACTOR][BLOCKING_FACTOR];
    // precalculate address
    int src_addr = (glb_block_i*BLOCKING_FACTOR+in_block_i+offset)*padded_num_vertex + (glb_block_j*BLOCKING_FACTOR+in_block_j);//
    int from_addr = (round*BLOCKING_FACTOR+in_block_i)*padded_num_vertex+(glb_block_j*BLOCKING_FACTOR + in_block_j);
    int to_addr = (glb_block_i*BLOCKING_FACTOR+in_block_i+offset)*padded_num_vertex+(round*BLOCKING_FACTOR + in_block_j);//

    // load value
    #pragma unroll
    for(int id=0;id<THREAD_LOAD_SAVE_NUM;id++){
        // init arr
        arr[id] = d_dist[src_addr + id];
        // load from and to
        from[in_block_i][in_block_j+id] = d_dist[from_addr + id];
        to[in_block_i][in_block_j+id] = d_dist[to_addr + id];
    }
    __syncthreads();
    for(int k=0;k<BLOCKING_FACTOR;k++){
        #pragma unroll
        for(int id=0;id<THREAD_LOAD_SAVE_NUM;id++){
            arr[id] = min(
                arr[id],
                to[in_block_i][k]+from[k][in_block_j+id]);
        }
    }
    #pragma unroll
    for(int id=0;id<THREAD_LOAD_SAVE_NUM;id++){
        d_dist[src_addr + id] = arr[id];
    }
}

__global__ void phase_3_a(int *d_dist, int padded_num_vertex, int round, int block_start_row){
    if(round == blockIdx.x || round == blockIdx.y + block_start_row) return;
    int glb_block_i = blockIdx.y + block_start_row;
    int glb_block_j = blockIdx.x;
    int in_block_i = threadIdx.y;
    int in_block_j = threadIdx.x * THREAD_LOAD_SAVE_NUM;
    int arr[THREAD_LOAD_SAVE_NUM];
    extern __shared__ int from[BLOCKING_FACTOR][BLOCKING_FACTOR];
    extern __shared__ int to[BLOCKING_FACTOR][BLOCKING_FACTOR];
    // precalculate address
    int src_addr = (glb_block_i*BLOCKING_FACTOR+in_block_i)*padded_num_vertex + (glb_block_j*BLOCKING_FACTOR+in_block_j);
    int from_addr = (round*BLOCKING_FACTOR+in_block_i)*padded_num_vertex+(glb_block_j*BLOCKING_FACTOR + in_block_j);
    int to_addr = (glb_block_i*BLOCKING_FACTOR+in_block_i)*padded_num_vertex+(round*BLOCKING_FACTOR + in_block_j);

    // load value
    #pragma unroll
    for(int id=0;id<THREAD_LOAD_SAVE_NUM;id++){
        // init arr
        arr[id] = d_dist[src_addr + id];
        // load from and to
        from[in_block_i][in_block_j+id] = d_dist[from_addr + id];
        to[in_block_i][in_block_j+id] = d_dist[to_addr + id];
    }
    __syncthreads();
    for(int k=0;k<BLOCKING_FACTOR;k++){
        #pragma unroll
        for(int id=0;id<THREAD_LOAD_SAVE_NUM;id++){
            arr[id] = min(
                arr[id],
                to[in_block_i][k]+from[k][in_block_j+id]);
        }
    }
    #pragma unroll
    for(int id=0;id<THREAD_LOAD_SAVE_NUM;id++){
        d_dist[src_addr + id] = arr[id];
    }
}

// [remote] pp23s80	—	41	249.29		0.32	0.17	0.17	0.17	0.17	0.17	0.17	0.17	0.22	0.22	0.22	0.17	0.17	0.17	0.22	0.32	0.32	0.47	0.47	0.97	1.17	1.77	2.32	2.92	3.62	4.53	5.33	6.18	7.18	7.98	9.18	10.54	11.94	13.62	16.35	17.50	18.96	22.61	23.87	27.43	28.92
void blocked_floyd_warshell(){
    int num_gpus;
    hipGetDeviceCount(&num_gpus);
    omp_set_num_threads(num_gpus);
    hipHostRegister(dist, padded_num_vertex * padded_num_vertex * sizeof(int), hipHostRegisterDefault);
    // allocate memory for dist in device
    int* d_dist[2];
    int y_offset = ((padded_num_vertex/BLOCKING_FACTOR) % 2 == 0)? padded_num_vertex/2: (padded_num_vertex/BLOCKING_FACTOR/2+1)*BLOCKING_FACTOR;

    #pragma omp parallel
    {
        num_blocked_square_row = padded_num_vertex / BLOCKING_FACTOR;
        num_blocked_square_col = padded_num_vertex / BLOCKING_FACTOR;
        dim3 basic_block(THREAD_X_DIM, THREAD_Y_DIM);
        dim3 phase_1_grid(1);
        dim3 phase_2_grid(num_blocked_square_row);
        dim3 phase_3_grid(num_blocked_square_row, y_offset/BLOCKING_FACTOR);
        
        unsigned int tid = omp_get_thread_num();
        hipSetDevice(tid);
        hipMalloc(&d_dist[tid], padded_num_vertex * padded_num_vertex * sizeof(int));
        hipMemcpy(d_dist[tid], dist, padded_num_vertex * padded_num_vertex * sizeof(int), hipMemcpyHostToDevice);

        for(int round=0;round<num_blocked_square_row;round++){
            phase_1      <<< phase_1_grid, basic_block>>> (d_dist[tid], padded_num_vertex, round);
            phase_2_fuse <<< phase_2_grid, basic_block>>> (d_dist[tid], padded_num_vertex, round);
            phase_3      <<< phase_3_grid, basic_block>>> (d_dist[tid], padded_num_vertex, round, y_offset*tid);
            hipDeviceSynchronize();
            #pragma omp barrier
            if(tid==1 && (round+1) < y_offset/BLOCKING_FACTOR){
                hipMemcpy(d_dist[1] + (round+1)*BLOCKING_FACTOR*padded_num_vertex, d_dist[0] + (round+1)*BLOCKING_FACTOR*padded_num_vertex, BLOCKING_FACTOR*padded_num_vertex*sizeof(int), hipMemcpyDeviceToDevice);
            }else if(tid==0 && (round+1)>=y_offset/BLOCKING_FACTOR){
                hipMemcpy(d_dist[0] + (round+1)*BLOCKING_FACTOR*padded_num_vertex, d_dist[1] + (round+1)*BLOCKING_FACTOR*padded_num_vertex, BLOCKING_FACTOR*padded_num_vertex*sizeof(int), hipMemcpyDeviceToDevice);
            }
        }
        if(tid==0)
            hipMemcpy(dist, d_dist[tid], y_offset*padded_num_vertex*sizeof(int), hipMemcpyDeviceToHost);
        else
            hipMemcpy(dist + y_offset*padded_num_vertex, d_dist[tid] + y_offset*padded_num_vertex, (padded_num_vertex-y_offset)*padded_num_vertex*sizeof(int), hipMemcpyDeviceToHost);
    }
}

int main(int argc, char* argv[]){
    assert(argc == 3);
    maxWidth = 5;
    char* input_file = argv[1];
    char* output_file = argv[2];
    // parse input file
    input(input_file);
    // blocked floyd-warshall
    blocked_floyd_warshell();
    // output file
    output(output_file);
    return 0;
}